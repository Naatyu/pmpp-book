
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK_X 2
#define THREADS_PER_BLOCK_Y 2

__global__ void squareMatMulKernel(float *M, float *N, float *P, int size)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < size && col < size)
    {
        float Pvalue = 0;
        for (int i = 0; i < size; i++)
        {
            Pvalue += M[row * size + i] * N[col + i * size];
        }
        P[row * size + col] = Pvalue;
    }
}

void squareMatMul(float *M, float *N, float *P, int size)
{
    // Compute size
    int allocSize = size * size * sizeof(float);

    // Allocate memory in device
    float *M_d;
    float *N_d;
    float *P_d;
    hipMalloc((void **)&M_d, allocSize);
    hipMalloc((void **)&N_d, allocSize);
    hipMalloc((void **)&P_d, allocSize);

    // Copy to device
    hipMemcpy(M_d, M, allocSize, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N, allocSize, hipMemcpyHostToDevice);
    hipMemcpy(P_d, P, allocSize, hipMemcpyHostToDevice);

    // Call kernel
    dim3 dimGrid(ceil(THREADS_PER_BLOCK_X), ceil(THREADS_PER_BLOCK_Y), 1);
    dim3 dimBlock(2, 2, 1);
    squareMatMulKernel<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, size);

    // Copy back P
    hipMemcpy(P, P_d, allocSize, hipMemcpyDeviceToHost);

    // Free
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

    return;
}

int main()
{
    const int size = 3;
    float M[] = {1, 1, 1, 2, 2, 2, 3, 3, 3};
    float N[] = {4, 4, 4, 5, 5, 5, 6, 6, 6};
    float P[size * size];

    squareMatMul(M, N, P, size);

    for (int i = 0; i < size; i++)
    {
        for (int k = 0; k < size; k++)
        {
            printf("%.2f ", P[i * size + k]);
        }
        printf("\n");
    }

    return 0;
}